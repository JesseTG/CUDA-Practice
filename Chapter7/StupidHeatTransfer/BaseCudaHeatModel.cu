#include "hip/hip_runtime.h"
#include "BaseCudaHeatModel.hpp"

__global__ static void fill_buffer_with_heaters(const Heater * __restrict__, size_t, float * __restrict__,
                                                uint2);

using namespace std;

BaseCudaHeatModel::BaseCudaHeatModel(uint2 d, const vector<Heater> &h)
    : BaseHeatModel(d, h), blocks(dim.x / 16, dim.y / 16), threads(16, 16) {
  handle(hipMalloc(&source, d.x * d.y * sizeof(*source)));
  handle(hipMalloc(&dest, d.x * d.y * sizeof(*dest)));
  handle(hipMalloc(&heaterCells, d.x * d.y * sizeof(*heaterCells)));

  handle(hipMemset(source, 0, d.x * d.y * sizeof(*source)));
  handle(hipMemset(dest, 0, d.x * d.y * sizeof(*dest)));
  handle(hipMemset(heaterCells, 0, d.x * d.y * sizeof(*heaterCells)));

  init_heaters();
}

BaseCudaHeatModel::~BaseCudaHeatModel() {
  handle(hipFree(source));
  handle(hipFree(dest));
  handle(hipFree(heaterCells));
}

void BaseCudaHeatModel::init_heaters() {
  CudaMemory<Heater> cudaHeaters(heaters.size());
  handle(hipMemcpy(cudaHeaters.ptr, heaters.data(),
                    heaters.size() * sizeof(Heater), hipMemcpyHostToDevice));

  fill_buffer_with_heaters<<<blocks, threads>>>(cudaHeaters.ptr, heaters.size(),
                                                heaterCells, dim);
  handle(hipGetLastError());
}

__global__ static void fill_buffer_with_heaters(const Heater * __restrict__ heaters,
                                                size_t numHeaters,
                                                float * __restrict__ heaterBuffer,
                                                uint2 dim) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  size_t offset = x + y * dim.x;

  for (size_t i = 0; i < numHeaters; ++i) {
    const Heater &heater = heaters[i];

    if (x >= heater.rect.x && y >= heater.rect.y &&
        x < (heater.rect.x + heater.rect.w) &&
        y < (heater.rect.y + heater.rect.h)) {
      heaterBuffer[offset] = heater.temperature;
    }
  }
}

void BaseCudaHeatModel::start_timing() {
  handle(hipEventRecord(frameStart.event, 0));
}

float BaseCudaHeatModel::stop_timing() {
  float duration = 0.0;
  handle(hipEventRecord(frameStop.event, 0));
  handle(hipEventSynchronize(frameStop.event));
  handle(hipEventElapsedTime(&duration, frameStart.event, frameStop.event));

  return duration;
}
